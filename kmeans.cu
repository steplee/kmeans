#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <algorithm>

#include "hip/hip_runtime_api.h"

#include <cstdlib>
#include <cstdio>

using namespace std;

const int K = K_FLAG; // 224
const int N = 32 * 1565; // 50080

struct point { float y; float x; };

point operator+(const point& a, const point& b) {
  return {a.y+b.y, a.x+b.x};
}
__host__ __device__ void operator+=(point& a, const point& b) {
  a.x += b.x;
  a.y += b.y;
}
void operator-=(point& a, const point& b) {
  a.x -= b.x;
  a.y -= b.y;
}
__device__ __host__ void operator/=(point& a, float f) {
  a.x /= f;
  a.y /= f;
}
__device__ __host__ point operator/(point& a, float f) {
  return {a.y/f, a.x/f};
}
ostream& operator<<(ostream& o, const point& a) {
  return o << "{" << a.y << ", " << a.x << "} ";
}

static float frand() {
  return static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
}

__host__ __device__ float dist(const point& a, const point& b) {
  //return sqrt( (a.y-b.y)*(a.y-b.y) + (a.y-b.y)(a.y-b.y) );
  return  (a.y-b.y)*(a.y-b.y) + (a.x-b.x)*(a.x-b.x);
}

__global__ void assign_pt(const point* pts, const point* mus, int* membership) {
  //int id = blockIdx.x*blockDim.x + threadIdx.x;
  int id = threadIdx.x;

  float best = 999999;

  for (int m=0; m<K; m++) {
    float d = dist(pts[id], mus[m]);
    if (d < best) {
      membership[id] = m;
      best = d;
    }
  }

  __threadfence_system();
}

__global__ void set_mu(point* mus, const point* pts, const int* membership) {
  //int id = blockIdx.x*blockDim.x + threadIdx.x;
  int id = threadIdx.x;

  // Having a local buffer may reduce writes to shared/global memory ??
  point local{0,0};
  int cnt = 0;

  for (int p=0; p<N; p++) {
    if (membership[p] == id) {
      local += pts[p];
      cnt++;
    }
  }

  mus[id] = local;

  if ( cnt > 0 )
    mus[id] /= ((float)cnt);

  __threadfence_system();
}


int main(int argc, char** argv) {

  point *pts, *mus;
  int *membership;

  pts = (point*) malloc( N * sizeof(point) );
  mus = (point*) malloc( K * sizeof(point) );
  membership = (int*) malloc( N * sizeof(int) );

  // Cuda setup.
  int dev = findCudaDevice(argc, (const char **)argv);

  srand(0);

  // Data.
  for (int i=0; i<N; i++) 
    pts[i] = {frand(), frand()},
    membership[i] = rand() % K;
  for (int i=0; i<K; i++) 
    mus[i] = {frand(), frand()};

  // TODO see if counting helps even with cuda.
  // I suspect it won't since we have to write to global memory sooo much.
  // Maybe write to shared & batch update?
  int mu_cnts[K];
  memset(mu_cnts, 0, sizeof(int)*K);

  point* d_mus;
  point* d_pts;
  int* d_membership;

  checkCudaErrors(hipMalloc((void**)&d_mus, sizeof(point)*K));
  checkCudaErrors(hipMalloc((void**)&d_pts, sizeof(point)*N));
  checkCudaErrors(hipMalloc((void**)&d_membership, sizeof(int)*N));
  checkCudaErrors(hipMemcpy(d_mus, mus, sizeof(point)*K, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_pts, pts, sizeof(point)*N, hipMemcpyHostToDevice));
  checkCudaErrors(hipMemcpy(d_membership, membership, sizeof(int)*N, hipMemcpyHostToDevice));

  // Run.
  for (int i=0; i<100; i++) {
    //cout << " (iter " << i << ")\n";

    // Phase 1: update point memberships
    assign_pt<<< 1, N >>>(d_pts, d_mus, d_membership);

    // Phase 2: update means
    set_mu<<< 1, K >>>(d_mus, d_pts, d_membership);

    //hipEventSynchronize(hipEventBlockingSync);
  }



  checkCudaErrors(hipMemcpy(mus, d_mus, sizeof(point)*K, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(pts, d_pts, sizeof(point)*N, hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(membership, d_membership, sizeof(int)*N, hipMemcpyDeviceToHost));

  // Eval.
  for (int m=210; m<K; m++) {
    int cnt = 0;
    for (int p=0; p<N; p++)
      if (membership[p] == m)
          cnt += 1;
    cout << " mu" << m << " has " << cnt << " points at " << mus[m] << ".\n";
  }

  return 0;
}
